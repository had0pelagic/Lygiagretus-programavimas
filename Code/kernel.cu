#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fstream>
#include <sstream>
#include <iomanip>
#include <iostream>
#include <hip/device_functions.h>
#include ""
#include "nlohmann/json.hpp"
#include "hip/hip_runtime.h"

#ifndef CUDACC_RTC
#define CUDACC_RTC
#endif

using namespace std;
using json = nlohmann::json;

const int THREADS = 3;
const int ARRAY_SIZE = 50;
const string IN_FILE = "1.json";
const string OUT_FILE = "out.txt";

class Student
{
public:
	char name[256];
	int year;
	double grade;
	int hash_sum;
};

void readJson(vector<Student> *students) {
	ifstream fr(IN_FILE);
	auto json = json::parse(fr);
	for (auto &element : json) {
		Student stud;
		string n = element["name"];
		strcpy(stud.name, n.c_str());
		stud.year = element["year"];
		stud.grade = element["grade"];
		stud.hash_sum = 0;
		students->push_back(stud);
	}
	fr.close();
}

__device__ int gpu_sum(size_t s) {
	int sum = 0;
	while (s > 0) {
		int num = s % 10;
		s /= 10;
		sum += num;
	}
	return sum;
}

__device__ void gpu_strcpy(char *dest, const char *src) {
	int i = 0;
	do {
		dest[i] = src[i];
	} while (src[i++] != 0);
}

__global__ void gpu_run(Student* device_students, int* device_array_size, int* device_slice_size, Student* device_results, int* device_result_count) {
	unsigned long start_index = *device_slice_size * threadIdx.x;
	unsigned long end_index;

	if (threadIdx.x == blockDim.x - 1)
		end_index = *device_array_size;
	else
		end_index = *device_slice_size * (threadIdx.x + 1);

	printf("///// THREAD: %d -- FROM: %d -- TO: %d \n", threadIdx.x, start_index, end_index);
	for (int i = start_index; i < end_index; i++) {
		Student stud;
		size_t bitshift = 2;
		size_t hash_name = ((size_t)device_students[i].name) >> bitshift;
		size_t hash_year = ((size_t)device_students[i].year) >> bitshift;
		size_t hash_grade = ((size_t)device_students[i].grade) >> bitshift;

		int sum_name = gpu_sum(hash_name);
		int sum_year = gpu_sum(hash_year);
		int sum_grade = gpu_sum(hash_grade);

		int all_sum = sum_grade + sum_year + sum_grade;
		if ((all_sum) % 2 == 0) {
			gpu_strcpy(stud.name, device_students[i].name);
			stud.hash_sum = all_sum;
			stud.year = device_students[i].year;
			stud.grade = device_students[i].grade;
			int index = atomicAdd(device_result_count, 1);
			device_results[index] = stud;
		}
	}
}

void output(vector<Student> student_vec, Student* student, int res_size) {
	ofstream wf(OUT_FILE);

	wf << string(39, '-') << "INPUT" << string(42, '-') << endl;
	wf << '|' << setw(20) << left << "Name" << setw(20) << left << "Year" << setw(15) << left << "Grade" << setw(28) << right << '|' << endl;
	wf << string(86, '-') << endl;
	for (size_t i = 0; i < ARRAY_SIZE; i++)
		wf << '|' << setw(20) << left << student_vec.at(i).name << setw(20) << left << student_vec.at(i).year << setw(15) << left << student_vec.at(i).grade << setw(28) << right << '|' << endl;

	wf << ARRAY_SIZE << endl;

	wf << string(40, '-') << "OUTPUT" << string(40, '-') << endl;
	wf << '|' << setw(20) << left << "Name" << setw(20) << left << "Year" << setw(18) << left << "Grade" << setw(13) << right << "Hash sum" << setw(18) << '|' << endl;
	wf << string(86, '-') << endl;

	for (size_t i = 0; i < res_size; i++)
		wf << '|' << setw(20) << left << student[i].name << setw(22) << left << student[i].year << setw(23) << left << student[i].grade << student[i].hash_sum << setw(22) << right << '|' << endl;
	wf << res_size << endl;
	wf << string(86, '-');

	wf.close();
}

int main() {
	vector<Student> student_vec;
	readJson(&student_vec);

	Student* students = &student_vec[0];
	Student results[ARRAY_SIZE];
	int slice_size = ARRAY_SIZE / THREADS;
	int result_count = 0;

	Student* device_students;
	Student* device_results;
	int* device_array_size;
	int* device_slice_size;
	int* device_result_count;

	hipMalloc((void**)&device_students, ARRAY_SIZE * sizeof(Student));
	hipMalloc((void**)&device_array_size, sizeof(int));
	hipMalloc((void**)&device_slice_size, sizeof(int));
	hipMalloc((void**)&device_result_count, sizeof(int));

	hipMalloc((void**)&device_results, ARRAY_SIZE * sizeof(Student));

	hipMemcpy(device_students, students, ARRAY_SIZE * sizeof(Student), hipMemcpyHostToDevice);
	hipMemcpy(device_array_size, &ARRAY_SIZE, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_slice_size, &slice_size, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_result_count, &result_count, sizeof(int), hipMemcpyHostToDevice);

	gpu_run << <1, THREADS >> > (device_students, device_array_size, device_slice_size, device_results, device_result_count);
	hipDeviceSynchronize();

	hipMemcpy(&results, device_results, ARRAY_SIZE * sizeof(Student), hipMemcpyDeviceToHost);
	int res_size = 0;
	hipMemcpy(&res_size, device_result_count, sizeof(int), hipMemcpyDeviceToHost);

	output(student_vec, results, res_size);

	return 0;
}